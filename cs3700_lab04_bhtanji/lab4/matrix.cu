#include "hip/hip_runtime.h"
// CS3700 Example matrix multpilcation using GPU

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 2
#define WIDTH 6

// Kernel function execute by the device (GPU)
__global__ void product (float *d_a, float *d_b, float *d_c, const int n) {
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   float sum = 0;
   if (row < n && col < n) {
      for (int i = 0 ; i<n ; ++i) {
         sum += d_a[row * n + i ] * d_b[i * n + col] ;
      }
      d_c[row * n + col] = sum;
   }
}

// Kernel for finding the sum of 2 matrices
__global__ void sum_m (float *d_a, float *d_b, float *d_c, const int n) {
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   float sum = 0;
   if(row < n && col < n){
      //add corresponding indices together
      sum = d_a[row*n+col] + d_b[row*n+col]; 

      //set that value to that square
      d_c[row * n + col] = sum; 

      //debug print location and result	
      //printf("d_a[%d] = %1.1f, d_b[%d] = %1.1f, sum[%d] = %1.1f \r\n", row*n+col, d_b[row*n+col], row*n+col, d_a[row*n+col], row*n+col, d_c[row*n+col]);
   }
   
}

// Utility function to print the input matrix
void printMatrix (float m[][WIDTH]) {
   int i, j;
   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j< WIDTH; ++j) {
         printf ("%d\t", (int)m[i][j]);
      }
      printf ("\n");
   }
}

// Main function execute by the host (CPU)
int main () {
   // host matrices
   float host_a[WIDTH][WIDTH],
         host_b[WIDTH][WIDTH],
         host_c[WIDTH][WIDTH];

   // device arrays
   float *device_a, *device_b, *device_c;

   int i, j;

   // initialize host matrices using random numbers
   time_t t;
   srand ((unsigned) time(&t));

   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j<WIDTH; j++) {
         host_a[i][j] = (float) (rand() % 50);
         host_b[i][j] = (float) (rand() % 50);
      }
   }

   printf ("Matrix A:\n");
   printMatrix (host_a);
   printf ("\n");

   printf ("Matrix B:\n");
   printMatrix (host_b);
   printf ("\n");

   // allocate device memory for input matrices
   size_t deviceSize = WIDTH * WIDTH * sizeof (float);
   hipMalloc ((void **) &device_a, deviceSize);
   hipMalloc ((void **) &device_b, deviceSize);

   // copy host matrices to device
   hipMemcpy (device_a, host_a, deviceSize, hipMemcpyHostToDevice );
   hipMemcpy (device_b, host_b, deviceSize, hipMemcpyHostToDevice );

   // allocate device memory to store computed result
   hipMalloc((void **) &device_c, deviceSize) ;

   /*
    * #define TILE_WIDTH 2
    * #define WIDTH 6
    */
   dim3 dimGrid (WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH);
   dim3 dimBlock (WIDTH, WIDTH);

   /****************** PRODUCT *********************/
   product<<<dimGrid, dimBlock>>> (device_a, device_b, device_c, WIDTH);
   // copy result from device back to host
   hipMemcpy (host_c, device_c, deviceSize, hipMemcpyDeviceToHost);

   // output the computed result matrix
   printf ("A x B: \n");
   printMatrix (host_c);

   /****************** SUM  *********************/
   sum_m<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, WIDTH);

   // copy result from device back to host
   hipMemcpy (host_c, device_c, deviceSize, hipMemcpyDeviceToHost);

   // output the computed result matrix
   printf ("A + B: \n");
   printMatrix (host_c);

   hipFree (device_a);
   hipFree (device_b);
   hipFree (device_c);
   return 0;
}
